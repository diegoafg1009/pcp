#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for dot product
__global__ void dotProduct(float *a, float *b, float *c, int size) {
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0;
    // Each thread computes part of the dot product
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // Store in shared memory
    cache[cacheIndex] = temp;
    
    // Synchronize threads
    __syncthreads();
    
    // Reduce within block
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    // Write result for this block to global memory
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main() {
    // Get device properties
    hipDeviceProp_t prop;
    int deviceId;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&prop, deviceId);
    
    // Print hardware specifications
    printf("CUDA Device Information:\n");
    printf("Device name: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimensions: (%d, %d, %d)\n", 
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total global memory: %.2f GB\n\n", 
           (float)prop.totalGlobalMem / (1024 * 1024 * 1024));

    // Set  size to 100 million elements
    size_t size = 100000000;  
    printf("\nVector size: %zu elements (%.2f GB per vector)\n", 
           size, (size * sizeof(float)) / (1024.0 * 1024.0 * 1024.0));
    
    float *a, *b;     // Host vectors
    float *d_a, *d_b, *d_c;  // Device vectors
    float result = 0;
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start timing total operation
    hipEventRecord(start);
    
    // Allocate host memory with pinned memory for better transfer performance
    hipHostMalloc(&a, size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&b, size * sizeof(float), hipHostMallocDefault);
    
    // Initialize vectors
    #pragma omp parallel for  // Optional: Use OpenMP for faster initialization
    for (size_t i = 0; i < size; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_c, ((size + 255)/256) * sizeof(float));
    
    // Copy data to device and measure transfer time
    hipEvent_t transfer_start, transfer_stop;
    hipEventCreate(&transfer_start);
    hipEventCreate(&transfer_stop);
    
    hipEventRecord(transfer_start);
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(transfer_stop);
    hipEventSynchronize(transfer_stop);
    
    float transfer_time = 0;
    hipEventElapsedTime(&transfer_time, transfer_start, transfer_stop);
    
    // Launch kernel with timing
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);
    
    hipEventRecord(kernel_start);
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    dotProduct<<<numBlocks, blockSize>>>(d_a, d_b, d_c, size);
    hipEventRecord(kernel_stop);
    hipEventSynchronize(kernel_stop);
    
    float kernel_time = 0;
    hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
    
    // Allocate host memory for partial results
    float *partial_c;
    hipHostMalloc(&partial_c, numBlocks * sizeof(float), hipHostMallocDefault);
    
    // Copy results back and measure
    hipEvent_t transfer_back_start, transfer_back_stop;
    hipEventCreate(&transfer_back_start);
    hipEventCreate(&transfer_back_stop);
    
    hipEventRecord(transfer_back_start);
    hipMemcpy(partial_c, d_c, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(transfer_back_stop);
    hipEventSynchronize(transfer_back_stop);
    
    float transfer_back_time = 0;
    hipEventElapsedTime(&transfer_back_time, transfer_back_start, transfer_back_stop);
    
    // Sum up partial results
    for (int i = 0; i < numBlocks; i++) {
        result += partial_c[i];
    }
    
    // Get total time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float total_time = 0;
    hipEventElapsedTime(&total_time, start, stop);
    
    // Print detailed timing results
    printf("\nResults and Timing:\n");
    printf("Dot product = %.1f\n", result);
    printf("Host to Device Transfer: %.2f ms\n", transfer_time);
    printf("Kernel Execution: %.2f ms\n", kernel_time);
    printf("Device to Host Transfer: %.2f ms\n", transfer_back_time);
    printf("Total Time: %.2f ms\n", total_time);
    
    // Calculate throughput
    float throughput = (2.0f * size * sizeof(float)) / (kernel_time * 1000000.0f);  // GB/s
    printf("Kernel Throughput: %.2f GB/s\n", throughput);
    
    // Cleanup
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(partial_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(transfer_start);
    hipEventDestroy(transfer_stop);
    hipEventDestroy(kernel_start);
    hipEventDestroy(kernel_stop);
    hipEventDestroy(transfer_back_start);
    hipEventDestroy(transfer_back_stop);
    
    return 0;
} 